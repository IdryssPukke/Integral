#define _USE_MATH_DEFINES

#include <iostream>
#include <math.h>
#include <cmath>
#include <Bits.h>
#include <numeric>

#include <hip/hip_runtime.h>
#include <nvfunctional>

#include <chrono>
#include <map>

using namespace std;

/* Pozwala na szybk� zmian� pomi�dzy double oraz float
* double - wi�ksza dok�adno��, wolniejsze dzia�anie
* float  - mniejsza dok�adno��, szybsze dzia�anie
*/
typedef double state;