#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include <iostream>
#include <math.h>
#include <cmath>
#include <Bits.h>
#include <numeric>

#include <hip/hip_runtime.h>
#include <nvfunctional>

#include <chrono>
#include <iomanip>

#include "gausse_legendre.cuh"
#include "rectangle.cuh"
#include "romberg.cuh"
#include "simpson38.cuh"
#include "simpson.cuh"
#include "Parameters.cu"

using namespace std;
using Clock = chrono::steady_clock;

//vectorAdd.cu


__device__ __host__  state f1(state x) {
	//return sin(x);
	return pow(exp(1.0),x);
	
	//przygotowana tablica, która zwraca tab[x]
}

void calculate_integral(state low_end, state high_end, state(*func)(state, state, int), string name, int i = 16) {
	
	state result = 0;
	state result_new = 0;

	for (int k = 0; i < INT_MAX; i *= 2, k++) {
		auto start = Clock::now();
		result_new = func(low_end, high_end, i);
		auto end = Clock::now();
		if (result_new - result < 10e-10 && k!=0) { 
			cout << setprecision(7) << fixed << name << ": \t" << result << "\t" << static_cast<chrono::duration<state>>(end - start).count() << endl << endl;
			break; 
		}
		else result = result_new;
	}	
}


int main() {

	state low_end = 0;
	state high_end =  M_PI;
	int steps = 100000;
	int N = 8;


	//first time to run CUDA, time not measured
	rectangle_method_CUDA(low_end, high_end, 1);

	calculate_integral(low_end, high_end, &rectangle_method, "rectangle", steps);
	calculate_integral(low_end, high_end, &rectangle_method_CUDA, "rectangleCUDA", steps);
	calculate_integral(low_end, high_end, &simpson_method,  "simpson", steps);
	calculate_integral(low_end, high_end, &simpson_method_CUDA, "simpsonCUDA", steps);
	calculate_integral(low_end, high_end, &simpson38_method, "simpson38", steps);
	calculate_integral(low_end, high_end, &simpson38_method_CUDA, "simpson38CUDA", steps);
	calculate_integral(low_end, high_end, &romberg_method, "romberg", N);
	calculate_integral(low_end, high_end, &romberg_method_CUDA_1, "rombergCUDA_1", 64);
	//calculate_integral(low_end, high_end, &romberg_method_CUDA_2, "rombergCUDA_2", N);

	
	//gauss_legendre
	auto start = Clock::now();
	cout << "gauss_legendre:" << gauss_legendre(128, f1, low_end, high_end) << " ";
	auto end = Clock::now();
	cout << static_cast<chrono::duration<state>>(end - start).count() << endl << endl;
	
	return 0;
}